#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "matrix_lib.h"

#define COLOR_CYAN "\033[0;36m"
#define COLOR_YELLOW "\033[1;33m"
#define COLOR_RESET "\033[0m"

#define DEVICE_DATASET_SIZE 1024000

#define THREADS_PER_BLOCK_LIMIT 1024
#define MAX_BLOCKS_PER_GRID_LIMIT 65535

static int threads_per_block = 256;
static int max_blocks_per_grid = 4096;

/** Determina o número de threads por bloco e o número máximo de blocos por grid */
void set_grid_size(int threads_per_block_param, int max_blocks_per_grid_param){
  if(threads_per_block_param < THREADS_PER_BLOCK_LIMIT && max_blocks_per_grid_param < MAX_BLOCKS_PER_GRID_LIMIT){
    threads_per_block = threads_per_block_param;
    max_blocks_per_grid = max_blocks_per_grid_param;

    return 1;
  }
  
  return 0;
}

/** Aloca uma matriz com a altura e a largura informadas. */
Matrix * create_matrix(int matrix_height, int matrix_width){
  
  Matrix * matrix = (Matrix *) malloc(sizeof(int) * 2 + sizeof(float) * (matrix_height*matrix_width) + (DEVICE_DATASET_SIZE * sizeof(float)));
  
  matrix->height = matrix_height;
  matrix->width = matrix_width;
  matrix->h_rows = (float *) malloc(matrix_height * matrix_width * sizeof(float));
  matrix->d_rows = (float *) malloc(DEVICE_DATASET_SIZE * sizeof(float))

  // check malloc memory allocation
  if (matrix->h_rows == NULL) { 
    printf("Error: malloc unable to allocate memory on host.");
    return 0;
  }

  hipError_t = hipMalloc(&(matrix->d_rows), matrix_height*matrix_width*sizeof(float));

  // check hipMalloc memory allocation
  if (hipError_t != hipSuccess) {
    printf("hipMalloc d_x returned error %s (code %d)\n",
    hipGetErrorString(hipError_t), hipError_t);
    return 0;
  }

  return matrix;
}


/** ------------READ MATRIX------------ **/

/** Preenche matriz com um arquivo .dat fornecido. */
int fill_matrix_with_file(FILE * file, Matrix * matrix){
  if(matrix == NULL){
    return 0;
  }

  int count = 0;
  float * vet = (float*) malloc((matrix->height * matrix->width) * sizeof(float));
  float vet_aux;

  for(int i = 0; i < matrix->height * matrix->width; i++){
		fread((void*) (&vet_aux), sizeof(vet_aux), 1, file);
		vet[count] = vet_aux;
		count++;
	}

  matrix->h_rows = vet;

  return 1;
}

/** Preenche matriz com um valor fornecido. */
int fill_matrix(float value, Matrix * matrix){
  if(matrix == NULL){
    printf("\nMatriz não declarada.\n");
    return 0;
  }

  printf("\nPreenchendo matriz...\n");

  for(int i = 0; i < matrix->height; i++){
    for (int j = 0; j < matrix->width; j++){
      matrix->rows[i * matrix->width + j] = value;
    }
  }

  return 1;
}


/** ----------SCALAR MATRIX MULT---------- **/

// Kernel function to scalar to array
__global__ 
void mult_scalar(int n, float *matrix_rows, float scalar_value)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
  if(index == 0){
    printf("\nblockDim.x=%d   gridDim.x%d   stride=%d\n", blockDim.x, gridDim.x, stride);
  }

  for (int i = index; i < n; i += stride) {
    matrix_rows[index] = matrix_rows[index] * scalar_value;
  }
}

/** Multiplica matriz por um valor fornecido utilizando a GPU. */
int scalar_matrix_mult(float scalar_value, Matrix * matrix){
  if(matrix == NULL){
    printf("\nMatriz não declarada.\n");
    return 0;
  }

  int matrix_size = matrix->height * matrix->width;

  int loop_limit = (matrix_size + DEVICE_DATASET_SIZE - 1) / DEVICE_DATASET_SIZE;
  int chunk_size = DEVICE_DATASET_SIZE;
  for(int count = 0; count < loop_limit; ++count){
    if(HOST_DATASET_SIZE % DEVICE_DATASET_SIZE != 0 && count == loop_limit - 1){
      chunk_size = HOST_DATASET_SIZE % DEVICE_DATASET_SIZE;
    }

    hipError_t = hipMemcpy(matrix->d_rows, matrix->h_rows+(count*DEVICE_DATASET_SIZE), chunk_size*sizeof(float), hipMemcpyHostToDevice);

    if (hipError_t != hipSuccess) {
      printf("hipMemcpy (h -> d) returned error %s (code %d), line(%d)\n", hipGetErrorString(hipError_t), hipError_t, __LINE__);
      return 0;
    }

    int blockSize = threads_per_block;
    int numBlocks = (chunk_size + blockSize - 1) / blockSize;
    if (numBlocks > max_blocks_per_grid) numBlocks = max_blocks_per_grid;

    mult_scalar<<<numBlocks, blockSize>>>(chunk_size, matrix->d_rows, scalar_value);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    hipError_t = hipMemcpy(matrix->h_rows+(count*chunk_size), matrix->d_rows, chunk_size*sizeof(float), hipMemcpyDeviceToHost);
  
    if (hipError_t != hipSuccess){
      printf("hipMemcpy (d -> h) returned error %s (code %d), line(%d)\n", hipGetErrorString(hipError_t), hipError_t, __LINE__);
      return 0;
    }
  }

  return 1;
}



/** ----------MATRIX MATRIX MULT---------- **/

/** Multiplica matriz A por matriz B de um valor fornecido de uma forma otimizada utilizando a GPU. */
int matrix_matrix_mult(Matrix * matrix_a, Matrix * matrix_b, Matrix * matrix_c){

  if(matrix_a == NULL || matrix_b == NULL){
    printf("\nUma ou duas das matrizes não declaradas.\n");
    return 0;
  }

  if(matrix_a->width != matrix_b->height){
    printf("\nA matriz A deve ter o número de colunas igual ao número de linhas da matriz B.\n");
    return 0;
  }

  for(int i = 0; i < matrix_a->height; i++){

    for(int j=0; j < matrix_a->width; j++){
      float position = matrix_a->rows[i * matrix_a->width + j];

       for(int k =0; k < matrix_b->width; k++){
         matrix_c->rows[i * matrix_c->width + k] += (position * matrix_b->rows[j+matrix_b->width + k]);
       }
    }

  }
  

  return 1;
}

/** ------------WRITE MATRIX------------ **/

/** Imprime a matriz fornecida */
int matrix_print(Matrix * matrix, char * nome){
  if(matrix == NULL){
    printf("\nMatriz não declarada.\n");
    return 0;
  }

  printf("\nImprimindo matriz %s para o console:\n",nome);


  for(int i = 0; i < matrix->height; i++){
    for (int j = 0; j < matrix->width; j++){
      printf("%.3f, ", matrix->rows[i * matrix->width + j]);
    }
    printf("\n");
  }

  return 1;
}

int write_matrix_to_file(FILE * file, Matrix * matrix){
  if(matrix == NULL){
    printf("\nMatriz não declarada.\n");
    return 0;
  }

  for(int i=0; i<matrix->height*matrix->width; i++){	
 		fwrite((void*)(&matrix->rows[i]), sizeof(matrix->rows[i]), 1, file);
	}

  return 1;
} 